#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void multiplication(int* a, int* b, int* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] * b[i];
}

__managed__ int a[256], b[256], c[256];

// name of main function changed (see one.cu)

int two_main() {
    for (int i = 0; i < 256; i++) {
        a[i] = i;
        b[i] = i * (i - (i - 5));
    }

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, 256 * sizeof(int));
    hipMalloc(&d_b, 256 * sizeof(int));
    hipMalloc(&d_c, 256 * sizeof(int));

    hipMemcpy(d_a, a, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 256 * sizeof(int), hipMemcpyHostToDevice);

    multiplication << <1, 256 >> > (d_a, d_b, d_c);

    hipMemcpy(c, d_c, 256 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 256; i++) {
        printf("%d * %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
